#include "CudaSparseMatrix.hpp"

#include <iostream>
#include <iomanip>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "SparseMatrixSumKernel.cuh"

#define CHECK_CUDA(call)                                        \
{                                                               \
    hipError_t err = call;                                     \
    if (err != hipSuccess) {                                   \
        std::cerr << "CUDA error in file " << __FILE__          \
                  << " at line " << __LINE__ << ": "            \
                  << hipGetErrorString(err) << std::endl;      \
        exit(EXIT_FAILURE);                                     \
    }                                                           \
}

#define CHECK_CUSPARSE(call)                                    \
{                                                               \
    hipsparseStatus_t err = call;                                \
    if (err != HIPSPARSE_STATUS_SUCCESS) {                       \
        std::cerr << "CUSPARSE error in file " << __FILE__      \
                  << " at line " << __LINE__ << ": "            \
                  << hipsparseGetErrorString(err) << std::endl;  \
        exit(EXIT_FAILURE);                                     \
    }                                                           \
}

const char* hipsparseGetErrorString(hipsparseStatus_t status) {
    switch (status) {
    case HIPSPARSE_STATUS_SUCCESS: return "HIPSPARSE_STATUS_SUCCESS";
    case HIPSPARSE_STATUS_NOT_INITIALIZED: return "HIPSPARSE_STATUS_NOT_INITIALIZED";
    case HIPSPARSE_STATUS_ALLOC_FAILED: return "HIPSPARSE_STATUS_ALLOC_FAILED";
    case HIPSPARSE_STATUS_INVALID_VALUE: return "HIPSPARSE_STATUS_INVALID_VALUE";
    case HIPSPARSE_STATUS_ARCH_MISMATCH: return "HIPSPARSE_STATUS_ARCH_MISMATCH";
    case HIPSPARSE_STATUS_MAPPING_ERROR: return "HIPSPARSE_STATUS_MAPPING_ERROR";
    case HIPSPARSE_STATUS_EXECUTION_FAILED: return "HIPSPARSE_STATUS_EXECUTION_FAILED";
    case HIPSPARSE_STATUS_INTERNAL_ERROR: return "HIPSPARSE_STATUS_INTERNAL_ERROR";
    case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED: return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
    case HIPSPARSE_STATUS_ZERO_PIVOT: return "HIPSPARSE_STATUS_ZERO_PIVOT";
    default: return "UNKNOWN CUSPARSE STATUS";
    }
}

CudaSparseMatrix::CudaSparseMatrix(int* I, int* J, float* V, int n, int nnz, SparseType sparseType, MemoryType memType): n_(n), nnz_(nnz)
{
    hipsparseHandle_t& cusparseHandle_ = CusparseHandle::getInstance();
    CHECK_CUDA(hipMalloc((void**)&d_csrOffsets_, (n_ + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_cols_, nnz_ * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_vals_, nnz_ * sizeof(float)));

    allocateAndCopy(I, J, V, sparseType, memType);

    CHECK_CUSPARSE(hipsparseCreateCsr(&matDescr_, n_, n_, nnz_,
        d_csrOffsets_, d_cols_, d_vals_,
        csr_row_ind_type_, csr_col_ind_type_,
        index_base_, valueType_));
}

CudaSparseMatrix::CudaSparseMatrix(const CudaSparseMatrix& other)
    : n_(other.n_), nnz_(other.nnz_) {
    hipsparseHandle_t& cusparseHandle_ = CusparseHandle::getInstance();
    CHECK_CUDA(hipMalloc((void**)&d_csrOffsets_, (n_ + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_cols_, nnz_ * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_vals_, nnz_ * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_csrOffsets_, other.d_csrOffsets_, (n_ + 1) * sizeof(int), hipMemcpyDeviceToDevice));
    CHECK_CUDA(hipMemcpy(d_cols_, other.d_cols_, nnz_ * sizeof(int), hipMemcpyDeviceToDevice));
    CHECK_CUDA(hipMemcpy(d_vals_, other.d_vals_, nnz_ * sizeof(float), hipMemcpyDeviceToDevice));

    CHECK_CUSPARSE(hipsparseCreateCsr(&matDescr_, n_, n_, nnz_,
        d_csrOffsets_, d_cols_, d_vals_,
        csr_row_ind_type_, csr_col_ind_type_,
        index_base_, valueType_));
}

CudaSparseMatrix::~CudaSparseMatrix() {
    CHECK_CUSPARSE(hipsparseDestroySpMat(matDescr_));
    CHECK_CUDA(hipFree(d_csrOffsets_));
    CHECK_CUDA(hipFree(d_cols_));
    CHECK_CUDA(hipFree(d_vals_));
}

void CudaSparseMatrix::updateData(const int* rows, const int* cols, const float* vals, int new_nnz, SparseType sparseType, MemoryType memType) {
    nnz_ = new_nnz;
    CHECK_CUDA(hipFree(d_cols_));
    CHECK_CUDA(hipFree(d_vals_));

    CHECK_CUDA(hipMalloc((void**)&d_cols_, nnz_ * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_vals_, nnz_ * sizeof(float)));

    allocateAndCopy(rows, cols, vals, sparseType, memType);
}

CudaDenseVector CudaSparseMatrix::dot(const float* d_vec)
{
    float alpha = 1.0f;
    float beta = 0.0f;
    hipsparseHandle_t& cusparseHandle_ = CusparseHandle::getInstance();
    CudaDenseVector result_vector = CudaDenseVector(n_);
    CudaDenseVector input_vector = CudaDenseVector(n_, d_vec, MemoryType::Device);
    
    size_t bufferSize = 0;
    void* dBuffer = nullptr;
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(cusparseHandle_,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matDescr_, input_vector.get(), &beta, result_vector.get(),
        valueType_, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
    
    CHECK_CUSPARSE(hipsparseSpMV(cusparseHandle_,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matDescr_, input_vector.get(), &beta, result_vector.get(),
        valueType_, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));
    
    CHECK_CUDA(hipFree(dBuffer));
    
    return result_vector;
}

void CudaSparseMatrix::allocateAndCopy(const int* rows, const int* cols, const float* vals, SparseType sparseType, MemoryType memType) {
    hipMemcpyKind copyType;

    if (memType == MemoryType::Host) {
        copyType = hipMemcpyHostToDevice;
        if (sparseType == SparseType::COO) {
            int* temp_d_rows;
            CHECK_CUDA(hipMalloc((void**)&temp_d_rows, nnz_ * sizeof(int)));
            CHECK_CUDA(hipMemcpy(temp_d_rows, rows, nnz_ * sizeof(int), copyType));
            rowsToCsr(temp_d_rows, d_csrOffsets_, n_, nnz_, sparseType);
            CHECK_CUDA(hipFree(temp_d_rows));
        }
        else {
            CHECK_CUDA(hipMemcpy(d_csrOffsets_, rows, (n_ + 1) * sizeof(int), copyType));
        }               
    }
    else {
        copyType = hipMemcpyDeviceToDevice;
        if (sparseType == SparseType::COO) {
            rowsToCsr(rows, d_csrOffsets_, n_, nnz_, sparseType);
        }
        else {
            CHECK_CUDA(hipMemcpy(d_csrOffsets_, rows, (n_ + 1) * sizeof(int), copyType));
        }
    }

    CHECK_CUDA(hipMemcpy(d_cols_, cols, nnz_ * sizeof(int), copyType));
    CHECK_CUDA(hipMemcpy(d_vals_, vals, nnz_ * sizeof(float), copyType));

}

void CudaSparseMatrix::rowsToCsr(const int* d_rows, int* d_csr_offset, int n, int nnz, SparseType sparseType)
{
    if (sparseType == SparseType::COO) {
        hipsparseHandle_t& handle = CusparseHandle::getInstance();
        hipsparseXcoo2csr(handle,
            d_rows,
            nnz,
            n,
            d_csr_offset,
            HIPSPARSE_INDEX_BASE_ZERO);
    }
    
}

float* CudaSparseMatrix::sumRows()
{
    hipsparseHandle_t& handle = CusparseHandle::getInstance();
    size_t bufferSize;
    void* dBuffer = nullptr;
    int* cscColPtr, *cscRowInd;
    float* cscVal, *diagonal;

    CHECK_CUDA(hipMalloc((void**)&cscColPtr, (n_ + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&cscRowInd, nnz_ * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&cscVal, nnz_ * sizeof(float)));

    CHECK_CUDA(hipMemset((void*)diagonal, 0, nnz_ * sizeof(float)));


    hipsparseCsr2cscEx2_bufferSize(handle, n_, n_, nnz_,
        d_vals_, d_csrOffsets_, d_cols_,
        cscVal, cscColPtr, cscRowInd,
        HIP_R_32F, HIPSPARSE_ACTION_NUMERIC,
        HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1,
        &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    // Conversion
    hipsparseCsr2cscEx2(handle, n_, n_, nnz_,
        d_vals_, d_csrOffsets_, d_cols_,
        cscVal, cscColPtr, cscRowInd,
        HIP_R_32F, HIPSPARSE_ACTION_NUMERIC,
        HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1,
        dBuffer);

    // Clean up
    hipFree(dBuffer);
    
    int blockSize = 512;
    int gridSize = (nnz_ + blockSize - 1) / blockSize;

    sum_axis << <gridSize, blockSize >> > (nnz_, cscRowInd, cscVal, diagonal);

    CHECK_CUDA(hipFree(cscColPtr));
    CHECK_CUDA(hipFree(cscRowInd));
    CHECK_CUDA(hipFree(cscVal));

    return diagonal;
}

float* CudaSparseMatrix::sumCols()
{
    hipsparseHandle_t& handle = CusparseHandle::getInstance();
    float* diagonal;

    CHECK_CUDA(hipMemset((void*)diagonal, 0, nnz_ * sizeof(float)));

    int blockSize = 512;
    int gridSize = (nnz_ + blockSize - 1) / blockSize;

    sum_axis << <gridSize, blockSize >> > (nnz_, d_cols_, d_vals_, diagonal);

    return diagonal;
}

float* CudaSparseMatrix::sum(int axis)
{
    return nullptr;
}

void CudaSparseMatrix::display()
{
    hipsparseHandle_t& handle = CusparseHandle::getInstance();
    float* d_denseMat;
    hipMalloc((void**)&d_denseMat, n_ * n_ * sizeof(float));

    // Create a dense matrix descriptor
    hipsparseDnMatDescr_t denseDescr;
    CHECK_CUSPARSE(hipsparseCreateDnMat(&denseDescr,
        n_, // number of rows
        n_, // number of columns
        n_, // leading dimension
        d_denseMat, // pointer to dense matrix data
        HIP_R_32F, // data type
        HIPSPARSE_ORDER_ROW)); // row-major order

    // Convert sparse matrix to dense matrix
    void* dBuffer = NULL;
    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseSparseToDense_bufferSize(handle,
        matDescr_,
        denseDescr,
        HIPSPARSE_SPARSETODENSE_ALG_DEFAULT,
        &bufferSize));

    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

    CHECK_CUSPARSE(hipsparseSparseToDense(handle,
        matDescr_,
        denseDescr,
        HIPSPARSE_SPARSETODENSE_ALG_DEFAULT,
        dBuffer));

    // Copy the dense matrix from device to host
    std::vector<float> h_denseMat(n_ * n_);
    CHECK_CUDA(hipMemcpy(h_denseMat.data(), d_denseMat, n_ * n_ * sizeof(float), hipMemcpyDeviceToHost));

    std::cout << std::fixed << std::setprecision(4); // Set precision to 2 decimal places
    std::cout << "Dense matrix:" << std::endl;
    for (int i = 0; i < n_; ++i) {
        for (int j = 0; j < n_; ++j) {
            std::cout << h_denseMat[i * n_ + j] << " ";
        }
        std::cout << std::endl;
    }

    // Clean up
    CHECK_CUDA(hipFree(d_denseMat));
    CHECK_CUDA(hipFree(dBuffer));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(denseDescr));
}

int CudaSparseMatrix::getNnz() const
{
    return nnz_;
}

int CudaSparseMatrix::size() const
{
    return n_;
}

CudaDenseVector::CudaDenseVector(int size, const float* V, MemoryType memType): size_(size)
{
    hipMemcpyKind copyType = memType == MemoryType::Host ? hipMemcpyHostToDevice : hipMemcpyDeviceToDevice;

    CHECK_CUDA(hipMalloc((void**)&d_data_, size_ * sizeof(float)));
    CHECK_CUDA(hipMemcpy(d_data_, V, size_ * sizeof(int), copyType));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecDescr_, size_, d_data_, HIP_R_32F));
}

CudaDenseVector::CudaDenseVector(int size)
{
    thrust::device_vector<float> input_vect = thrust::device_vector<float>(size_, 0.0f);
    CudaDenseVector(size_, thrust::raw_pointer_cast(input_vect.data()), MemoryType::Device);
}

CudaDenseVector::~CudaDenseVector()
{
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecDescr_));
    CHECK_CUDA(hipFree(d_data_));
}

int CudaDenseVector::size() const
{
    return size_;
}

float* CudaDenseVector::data() const
{
    return d_data_;
}

hipsparseDnVecDescr_t CudaDenseVector::get() const
{
    return vecDescr_;
}
