#include "CudaSparseMatrix.hpp"

#include <iostream>
#include <iomanip>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>


const char* hipsparseGetErrorString(hipsparseStatus_t status) {
    switch (status) {
    case HIPSPARSE_STATUS_SUCCESS: return "HIPSPARSE_STATUS_SUCCESS";
    case HIPSPARSE_STATUS_NOT_INITIALIZED: return "HIPSPARSE_STATUS_NOT_INITIALIZED";
    case HIPSPARSE_STATUS_ALLOC_FAILED: return "HIPSPARSE_STATUS_ALLOC_FAILED";
    case HIPSPARSE_STATUS_INVALID_VALUE: return "HIPSPARSE_STATUS_INVALID_VALUE";
    case HIPSPARSE_STATUS_ARCH_MISMATCH: return "HIPSPARSE_STATUS_ARCH_MISMATCH";
    case HIPSPARSE_STATUS_MAPPING_ERROR: return "HIPSPARSE_STATUS_MAPPING_ERROR";
    case HIPSPARSE_STATUS_EXECUTION_FAILED: return "HIPSPARSE_STATUS_EXECUTION_FAILED";
    case HIPSPARSE_STATUS_INTERNAL_ERROR: return "HIPSPARSE_STATUS_INTERNAL_ERROR";
    case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED: return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
    case HIPSPARSE_STATUS_ZERO_PIVOT: return "HIPSPARSE_STATUS_ZERO_PIVOT";
    default: return "UNKNOWN CUSPARSE STATUS";
    }
}

CudaSparseMatrix::CudaSparseMatrix(int* I, int* J, float* V, int n, int nnz, SparseType sparseType, MemoryType memType): n_(n), nnz_(nnz)
{
    hipsparseHandle_t& cusparseHandle_ = CusparseHandle::getInstance();
    CHECK_CUDA(hipMalloc((void**)&d_csrOffsets_, (n_ + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_cols_, nnz_ * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_vals_, nnz_ * sizeof(float)));

    allocateAndCopy(I, J, V, sparseType, memType);

    CHECK_CUSPARSE(hipsparseCreateCsr(&matDescr_, n_, n_, nnz_,
        d_csrOffsets_, d_cols_, d_vals_,
        csr_row_ind_type_, csr_col_ind_type_,
        index_base_, valueType_));
}

CudaSparseMatrix::CudaSparseMatrix(const CudaSparseMatrix& other)
    : n_(other.n_), nnz_(other.nnz_), matDescr_(nullptr) {
    hipsparseHandle_t& cusparseHandle_ = CusparseHandle::getInstance();
    CHECK_CUDA(hipMalloc((void**)&d_csrOffsets_, (n_ + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_cols_, nnz_ * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_vals_, nnz_ * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_csrOffsets_, other.d_csrOffsets_, (n_ + 1) * sizeof(int), hipMemcpyDeviceToDevice));
    CHECK_CUDA(hipMemcpy(d_cols_, other.d_cols_, nnz_ * sizeof(int), hipMemcpyDeviceToDevice));
    CHECK_CUDA(hipMemcpy(d_vals_, other.d_vals_, nnz_ * sizeof(float), hipMemcpyDeviceToDevice));

    CHECK_CUSPARSE(hipsparseCreateCsr(&matDescr_, n_, n_, nnz_,
        d_csrOffsets_, d_cols_, d_vals_,
        csr_row_ind_type_, csr_col_ind_type_,
        index_base_, valueType_));
}

CudaSparseMatrix::~CudaSparseMatrix() {
    clear();
}

void CudaSparseMatrix::updateData(const int* rows, const int* cols, const float* vals, int new_nnz, SparseType sparseType, MemoryType memType) {
    nnz_ = new_nnz;
    CHECK_CUDA(hipFree(d_cols_));
    CHECK_CUDA(hipFree(d_vals_));
    CHECK_CUSPARSE(hipsparseDestroySpMat(matDescr_));

    CHECK_CUDA(hipMalloc((void**)&d_cols_, nnz_ * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_vals_, nnz_ * sizeof(float)));

    allocateAndCopy(rows, cols, vals, sparseType, memType);

    CHECK_CUSPARSE(hipsparseCreateCsr(&matDescr_, n_, n_, nnz_,
        d_csrOffsets_, d_cols_, d_vals_,
        csr_row_ind_type_, csr_col_ind_type_,
        index_base_, valueType_));
}

bool* CudaSparseMatrix::zero_elements_in_vector(const float* input_vect, int& zero_sum, int n) {
    hipsparseHandle_t& handle = CusparseHandle::getInstance();
    bool* zero_elements_vect;
    int* d_zero_sum;

    zero_sum = 0;

    // Allocate memory on the device
    CHECK_CUDA(hipMalloc((void**)&zero_elements_vect, n * sizeof(bool)));
    CHECK_CUDA(hipMalloc((void**)&d_zero_sum, sizeof(int)));

    // Initialize memory
    CHECK_CUDA(hipMemset(zero_elements_vect, 0, n * sizeof(bool)));
    CHECK_CUDA(hipMemset(d_zero_sum, 0, sizeof(int)));

    int gridSize = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    zero_elements << <gridSize, BLOCK_SIZE >> > (input_vect, zero_elements_vect, d_zero_sum, n);
    CHECK_CUDA(hipDeviceSynchronize());

    // Copy the result back to host
    CHECK_CUDA(hipMemcpy(&zero_sum, d_zero_sum, sizeof(int), hipMemcpyDeviceToHost));

    // Free device memory
    CHECK_CUDA(hipFree(d_zero_sum));

    return zero_elements_vect;
}

void CudaSparseMatrix::fill_diagonal(const float* diagonal_vect)
{
    int nnz_sum = 0;
    int zero_sum = 0;
    int diag_nnz = n_;
    int resize_n = diag_nnz;
    // TODO: Flip to non zero vector and copy only the non zero elements to new vector
    bool* zeros_in_diag_sum = zero_elements_in_vector(diagonal_vect, zero_sum, n_);
    bool* non_zero = non_zero_diagonal(nnz_sum);
    
    bool* h_non_zero = new bool[n_];
    CHECK_CUDA(hipMemcpy(h_non_zero, zeros_in_diag_sum, n_ * sizeof(bool), hipMemcpyDeviceToHost));

    for (int i = 0; i < n_; i++)
    {
        std::cout << "zero_in_diag_" << i << ": " << h_non_zero[i] << std::endl;
    }
    
    diag_nnz -= zero_sum;
    resize_n = diag_nnz - nnz_sum;

    int* original_I, * new_I, * new_J;
    float* new_V;

    CHECK_CUDA(hipMalloc((void**)&original_I, (nnz_) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&new_I, (nnz_ + resize_n) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&new_J, (nnz_ + resize_n) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&new_V, (nnz_ + resize_n) * sizeof(float)));

    csrTorows(d_csrOffsets_, original_I, n_, nnz_, SparseType::CSR);

    CHECK_CUDA(hipMemcpy(new_I, original_I, nnz_ * sizeof(int), hipMemcpyDeviceToDevice));
    CHECK_CUDA(hipMemcpy(new_J, d_cols_, nnz_ * sizeof(int), hipMemcpyDeviceToDevice));
    CHECK_CUDA(hipMemcpy(new_V, d_vals_, nnz_ * sizeof(float), hipMemcpyDeviceToDevice));

    int gridSize = ((nnz_+n_)+BLOCK_SIZE - 1) / BLOCK_SIZE;
    set_diagonal << <gridSize, BLOCK_SIZE >> > (new_I, new_J, new_V, non_zero, diagonal_vect, nnz_, resize_n);
    CHECK_CUDA(hipDeviceSynchronize());
    thrust::device_ptr<int> dev_I(new_I);
    thrust::device_ptr<int> dev_J(new_J);
    thrust::device_ptr<float> dev_V(new_V);

    // First, sort by the secondary key (J) using stable sort
    thrust::stable_sort_by_key(dev_J, dev_J + (nnz_ + resize_n), thrust::make_zip_iterator(thrust::make_tuple(dev_I, dev_V)));

    // Then, sort by the primary key (I) using stable sort to maintain the order of the secondary key
    thrust::stable_sort_by_key(dev_I, dev_I + (nnz_ + resize_n), thrust::make_zip_iterator(thrust::make_tuple(dev_J, dev_V)));

    float* h_new_I = new float[nnz_+resize_n];
    CHECK_CUDA(hipMemcpy(h_new_I, new_V, (nnz_ + resize_n) * sizeof(float), hipMemcpyDeviceToHost));

    for (int i = 0; i < nnz_ + resize_n; i++)
    {
        std::cout << "Copied V_" << i << ": " << h_new_I[i] << std::endl;
    }

    updateData(new_I, new_J, new_V, nnz_ + resize_n, SparseType::COO, MemoryType::Device);

    CHECK_CUDA(hipFree(original_I));
    CHECK_CUDA(hipFree(new_I));
    CHECK_CUDA(hipFree(new_J));
    CHECK_CUDA(hipFree(new_V));

    std::cout << "Total non zero elements on the diagonal: " << nnz_sum << std::endl;
}

bool* CudaSparseMatrix::non_zero_diagonal(int& nnz_diag_sum)
{
    hipsparseHandle_t& handle = CusparseHandle::getInstance();
    bool* nnz_diag;
    int* I;
    int* d_nnz_diag_sum;
    nnz_diag_sum = 0;

    // Allocate memory on the device
    CHECK_CUDA(hipMalloc((void**)&d_nnz_diag_sum, sizeof(int)));

    // Initialize memory
    CHECK_CUDA(hipMemset(d_nnz_diag_sum, 0, sizeof(int)));

    CHECK_CUDA(hipMalloc((void**)&I, nnz_ * sizeof(int)));
    hipsparseXcsr2coo(handle,
        d_csrOffsets_,
        nnz_,
        n_,
        I,
        HIPSPARSE_INDEX_BASE_ZERO);


    CHECK_CUDA(hipMalloc((void**)&nnz_diag, n_ * sizeof(bool)));
    CHECK_CUDA(hipMemset(nnz_diag, 0, n_));
    int gridSize = (nnz_ + BLOCK_SIZE - 1) / BLOCK_SIZE;
    non_zero_elements << <gridSize, BLOCK_SIZE >> > (I, d_cols_, nnz_diag, d_nnz_diag_sum, nnz_);

    CHECK_CUDA(hipMemcpy(&nnz_diag_sum, d_nnz_diag_sum, sizeof(int), hipMemcpyDeviceToHost));

    // Free device memory
    CHECK_CUDA(hipFree(d_nnz_diag_sum));
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipFree(I));

    return nnz_diag;
}

CudaDenseVector CudaSparseMatrix::dot(const float* d_vec)
{
    float alpha = 1.0f;
    float beta = 0.0f;
    hipsparseHandle_t& cusparseHandle_ = CusparseHandle::getInstance();
    CudaDenseVector result_vector = CudaDenseVector(n_);
    CudaDenseVector input_vector = CudaDenseVector(n_, d_vec, MemoryType::Device);
    
    size_t bufferSize = 0;
    void* dBuffer = nullptr;
    CHECK_CUSPARSE(hipsparseSpMV_bufferSize(cusparseHandle_,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matDescr_, input_vector.get(), &beta, result_vector.get(),
        valueType_, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
    
    CHECK_CUSPARSE(hipsparseSpMV(cusparseHandle_,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matDescr_, input_vector.get(), &beta, result_vector.get(),
        valueType_, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));
    
    CHECK_CUDA(hipFree(dBuffer));
    
    return result_vector;
}

void CudaSparseMatrix::multiply(float value)
{
    hipsparseHandle_t& handle = CusparseHandle::getInstance();
    // Set scaling factors
    const float beta = 0.0f;
    size_t bufferSize = 0;

    hipsparseMatDescr_t input_desc;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&input_desc));

    // Create matrix descriptor for the result matrix C
    hipsparseMatDescr_t result_desc;
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&result_desc));

    // Get buffer size for the operation
    CHECK_CUSPARSE(hipsparseScsrgeam2_bufferSizeExt(handle,
        n_, n_,
        &value, input_desc, nnz_,
        d_vals_,
        d_csrOffsets_,
        d_cols_,
        &beta, nullptr, nnz_,
        nullptr,
        nullptr,
        nullptr,
        result_desc,
        d_vals_,
        d_csrOffsets_,
        d_cols_,
        &bufferSize));

    void* dBuffer;
    hipMalloc(&dBuffer, bufferSize);

    // Perform the scaling operation
    hipsparseScsrgeam2(handle,
        n_, n_,
        &value, input_desc, nnz_,
        d_vals_,
        d_csrOffsets_,
        d_cols_,
        &beta, input_desc, nnz_,
        d_vals_,
        d_csrOffsets_,
        d_cols_,
        result_desc,
        d_vals_,
        d_csrOffsets_,
        d_cols_,
        dBuffer);
    // Clean up
    hipFree(dBuffer);
    hipsparseDestroyMatDescr(input_desc);
    hipsparseDestroyMatDescr(result_desc);

}

void CudaSparseMatrix::allocateAndCopy(const int* rows, const int* cols, const float* vals, SparseType sparseType, MemoryType memType) {
    hipMemcpyKind copyType;

    if (memType == MemoryType::Host) {
        copyType = hipMemcpyHostToDevice;
        if (sparseType == SparseType::COO) {
            int* temp_d_rows;
            CHECK_CUDA(hipMalloc((void**)&temp_d_rows, nnz_ * sizeof(int)));
            CHECK_CUDA(hipMemcpy(temp_d_rows, rows, nnz_ * sizeof(int), copyType));
            rowsToCsr(temp_d_rows, d_csrOffsets_, n_, nnz_, sparseType);
            CHECK_CUDA(hipFree(temp_d_rows));
        }
        else {
            CHECK_CUDA(hipMemcpy(d_csrOffsets_, rows, (n_ + 1) * sizeof(int), copyType));
        }               
    }
    else {
        copyType = hipMemcpyDeviceToDevice;
        if (sparseType == SparseType::COO) {
            rowsToCsr(rows, d_csrOffsets_, n_, nnz_, sparseType);
        }
        else {
            CHECK_CUDA(hipMemcpy(d_csrOffsets_, rows, (n_ + 1) * sizeof(int), copyType));
        }
    }

    CHECK_CUDA(hipMemcpy(d_cols_, cols, nnz_ * sizeof(int), copyType));
    CHECK_CUDA(hipMemcpy(d_vals_, vals, nnz_ * sizeof(float), copyType));

}

void CudaSparseMatrix::rowsToCsr(const int* d_rows, int* d_csr_offset, int n, int nnz, SparseType sparseType)
{
    if (sparseType == SparseType::COO) {
        hipsparseHandle_t& handle = CusparseHandle::getInstance();
        hipsparseXcoo2csr(handle,
            d_rows,
            nnz,
            n,
            d_csr_offset,
            HIPSPARSE_INDEX_BASE_ZERO);
    }
    
}

void CudaSparseMatrix::csrTorows(const int* d_csr_offset, int* d_rows, int n, int nnz, SparseType sparseType)
{
    if (sparseType == SparseType::CSR) {
        hipsparseHandle_t& handle = CusparseHandle::getInstance();
        hipsparseXcsr2coo(handle,
            d_csr_offset,
            nnz,
            n,
            d_rows,
            HIPSPARSE_INDEX_BASE_ZERO);
    }
}

float* CudaSparseMatrix::sumRows()
{
    hipsparseHandle_t& handle = CusparseHandle::getInstance();
    size_t bufferSize;
    void* dBuffer = nullptr;
    int* cscColPtr, *cscRowInd;
    float* cscVal, *diagonal;

    CHECK_CUDA(hipMalloc((void**)&cscColPtr, (n_ + 1) * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&cscRowInd, nnz_ * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&cscVal, nnz_ * sizeof(float)));

    CHECK_CUDA(hipMalloc((void**)&diagonal, n_ * sizeof(float)));
    CHECK_CUDA(hipMemset((void*)diagonal, 0, n_ * sizeof(float)));


    hipsparseCsr2cscEx2_bufferSize(handle, n_, n_, nnz_,
        d_vals_, d_csrOffsets_, d_cols_,
        cscVal, cscColPtr, cscRowInd,
        HIP_R_32F, HIPSPARSE_ACTION_NUMERIC,
        HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1,
        &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    // Conversion
    hipsparseCsr2cscEx2(handle, n_, n_, nnz_,
        d_vals_, d_csrOffsets_, d_cols_,
        cscVal, cscColPtr, cscRowInd,
        HIP_R_32F, HIPSPARSE_ACTION_NUMERIC,
        HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1,
        dBuffer);

    // Clean up
    hipFree(dBuffer);
    
    int gridSize = (nnz_ + BLOCK_SIZE - 1) / BLOCK_SIZE;

    sum_axis << <gridSize, BLOCK_SIZE >> > (nnz_, cscRowInd, cscVal, diagonal);

    CHECK_CUDA(hipFree(cscColPtr));
    CHECK_CUDA(hipFree(cscRowInd));
    CHECK_CUDA(hipFree(cscVal));

    return diagonal;
}

float* CudaSparseMatrix::sumCols()
{
    hipsparseHandle_t& handle = CusparseHandle::getInstance();
    float* diagonal;

    CHECK_CUDA(hipMalloc((void**)&diagonal, n_ * sizeof(float)));
    CHECK_CUDA(hipMemset((void*)diagonal, 0, n_ * sizeof(float)));

    int blockSize = 512;
    int gridSize = (nnz_ + blockSize - 1) / blockSize;

    sum_axis << <gridSize, blockSize >> > (nnz_, d_cols_, d_vals_, diagonal);

    return diagonal;
}

float* CudaSparseMatrix::sum(int axis)
{
    if (axis == 0) {
        return sumRows();
    }

    if (axis == 1) {
        return sumCols();
    }

    return nullptr;
}

void CudaSparseMatrix::display()
{
    hipsparseHandle_t& handle = CusparseHandle::getInstance();
    float* d_denseMat;
    hipMalloc((void**)&d_denseMat, n_ * n_ * sizeof(float));

    // Create a dense matrix descriptor
    hipsparseDnMatDescr_t denseDescr;
    CHECK_CUSPARSE(hipsparseCreateDnMat(&denseDescr,
        n_, // number of rows
        n_, // number of columns
        n_, // leading dimension
        d_denseMat, // pointer to dense matrix data
        HIP_R_32F, // data type
        HIPSPARSE_ORDER_ROW)); // row-major order

    // Convert sparse matrix to dense matrix
    void* dBuffer = NULL;
    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseSparseToDense_bufferSize(handle,
        matDescr_,
        denseDescr,
        HIPSPARSE_SPARSETODENSE_ALG_DEFAULT,
        &bufferSize));

    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

    CHECK_CUSPARSE(hipsparseSparseToDense(handle,
        matDescr_,
        denseDescr,
        HIPSPARSE_SPARSETODENSE_ALG_DEFAULT,
        dBuffer));

    // Copy the dense matrix from device to host
    std::vector<float> h_denseMat(n_ * n_);
    CHECK_CUDA(hipMemcpy(h_denseMat.data(), d_denseMat, n_ * n_ * sizeof(float), hipMemcpyDeviceToHost));

    std::cout << std::fixed << std::setprecision(4); // Set precision to 2 decimal places
    std::cout << "Dense matrix:" << std::endl;
    for (int i = 0; i < n_; ++i) {
        for (int j = 0; j < n_; ++j) {
            std::cout << std::setw(7) << h_denseMat[i * n_ + j] << " ";
        }
        std::cout << std::endl;
    }

    // Clean up
    CHECK_CUDA(hipFree(d_denseMat));
    CHECK_CUDA(hipFree(dBuffer));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(denseDescr));
}

int CudaSparseMatrix::getNnz() const
{
    return nnz_;
}

int CudaSparseMatrix::size() const
{
    return n_;
}

void CudaSparseMatrix::clear()
{
    if (d_csrOffsets_) {
        CHECK_CUDA(hipFree(d_csrOffsets_));
        std::cout << "d_csrOffsets_ cleared" << std::endl;
        d_csrOffsets_ = nullptr;
    }
    if (d_cols_) {
        CHECK_CUDA(hipFree(d_cols_));
        std::cout << "d_cols_ cleared" << std::endl;
        d_cols_ = nullptr;
    }
    if (d_vals_) {
        CHECK_CUDA(hipFree(d_vals_));
        std::cout << "d_vals_ cleared" << std::endl;
        d_vals_ = nullptr;
    }
    if (matDescr_) {
        CHECK_CUSPARSE(hipsparseDestroySpMat(matDescr_));
        std::cout << "matDescr_ cleared" << std::endl;
        matDescr_ = nullptr;
    }

    nnz_ = 0;
    n_ = 0;
}

CudaDenseVector::CudaDenseVector(int size, const float* V, MemoryType memType): size_(size)
{
    hipMemcpyKind copyType = memType == MemoryType::Host ? hipMemcpyHostToDevice : hipMemcpyDeviceToDevice;

    CHECK_CUDA(hipMalloc((void**)&d_data_, size_ * sizeof(float)));
    CHECK_CUDA(hipMemcpy(d_data_, V, size_ * sizeof(int), copyType));
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecDescr_, size_, d_data_, HIP_R_32F));
}

CudaDenseVector::CudaDenseVector(int size)
{
    thrust::device_vector<float> input_vect = thrust::device_vector<float>(size_, 0.0f);
    CudaDenseVector(size_, thrust::raw_pointer_cast(input_vect.data()), MemoryType::Device);
}

CudaDenseVector::~CudaDenseVector()
{
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecDescr_));
    CHECK_CUDA(hipFree(d_data_));
}

int CudaDenseVector::size() const
{
    return size_;
}

float* CudaDenseVector::data() const
{
    return d_data_;
}

hipsparseDnVecDescr_t CudaDenseVector::get() const
{
    return vecDescr_;
}
